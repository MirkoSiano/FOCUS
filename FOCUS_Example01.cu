#include <iostream>
#include <math.h>
#include "FOCUS_Class.cuh"

using namespace std;

int main(int argc, char **argv){

	// pointer to FOCUS class
	FOCUS *focus;
	focus = new FOCUS;

	// configure focus
	focus->readConfigFiles();

	// allocate phaseSpace
	focus->allocatePhaseSpace();

	// fill phase space
	focus->fillPhaseSpace();

	// copy phase space on GPU memory
	focus->copyPhaseSpaceFromHostToDevice();

	// compute 1D coherence profile
	focus->coherence1D();

	// save
	focus->saveCoherence1D();
	focus->saveParameters();

	// free memory
	focus->freePhaseSpace();

	// closing program
	delete focus;

	return 0;
}